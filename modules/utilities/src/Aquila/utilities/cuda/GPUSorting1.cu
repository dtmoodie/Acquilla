#include <Aquila/core/detail/Export.hpp>
#include <Aquila/utilities/thrust/thrust_interop.hpp>
#include "Aquila/utilities/cuda/GPUSortingPriv.hpp"
namespace cv
{
    namespace cuda
    {
        namespace detail
        {
            template AQUILA_EXPORTS void sortAscending<ushort>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortDescending<ushort>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortAscendingEachRow<ushort>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortDescendingEachRow<ushort>(cv::cuda::GpuMat&, hipStream_t);

            template AQUILA_EXPORTS void sortAscending<short>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortDescending<short>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortAscendingEachRow<short>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortDescendingEachRow<short>(cv::cuda::GpuMat&, hipStream_t);

            template AQUILA_EXPORTS void sortAscending<int>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortDescending<int>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortAscendingEachRow<int>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortDescendingEachRow<int>(cv::cuda::GpuMat&, hipStream_t);
        }
    }
}
