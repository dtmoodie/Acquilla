#include <Aquila/core/detail/Export.hpp>
#include <Aquila/utilities/thrust/thrust_interop.hpp>
#include <Aquila/utilities/cuda/GPUSortingPriv.hpp>
namespace cv
{
    namespace cuda
    {
        namespace detail
        {
            template AQUILA_EXPORTS void sortAscending<float>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortDescending<float>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortAscendingEachRow<float>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortDescendingEachRow<float>(cv::cuda::GpuMat&, hipStream_t);

            template AQUILA_EXPORTS void sortAscending<double>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortDescending<double>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortAscendingEachRow<double>(cv::cuda::GpuMat&, hipStream_t);
            template AQUILA_EXPORTS void sortDescendingEachRow<double>(cv::cuda::GpuMat&, hipStream_t);
        }
    }
}
